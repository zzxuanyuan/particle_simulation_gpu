#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include "common.h"


#define MAX_ENTRIES 100 

//
//  benchmarking program
//
int main( int argc, char **argv )
{
    int n[MAX_ENTRIES],i,count=0,num;
    double t[MAX_ENTRIES],slope[MAX_ENTRIES-1],ss[MAX_ENTRIES],grade,ssgrade,ss_avg;
    double lt[MAX_ENTRIES],ln[MAX_ENTRIES],b2,sx=0.0,sx2=0.0,sxy=0.0,sy=0.0;

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help \n" );
        printf( "-s <filename> to specify name of summary file \n");
        printf( "-v to specify what to autograde (gpu) \n" );
        return 0;
    }
    
    char *savename = read_string( argc, argv, "-s", NULL );
    FILE *fread = savename ? fopen( savename, "r" ) : NULL;

    char *autoname = read_string( argc, argv, "-v", NULL );
     
    if (strcmp(autoname,"gpu")==0){
      if(fread) 
        while( fscanf (fread,"%d %lf",&n[count],&t[count]) != EOF ) 
          count++;

      num = count/2;     

      for (i=0; i<num-1;i++) {
        slope [i] = ( log(t[i+1]) - log(t[i]) ) / ( log(n[i+1]*1.0) - log(n[i]*1.0) );
      }
      for (i=0; i<num; i++) {
        lt[i] = log(t[i]);
        ln[i] = log(n[i]*1.0);
      }
      for (i=0; i<num; i++) {
        sx += ln[i];
        sy += lt[i];
        sxy += ln[i]*lt[i];
        sx2 += ln[i]*ln[i];
      }

      b2 = (sxy - (sx * sy)/ (num * 1.0) ) / (sx2 - (sx * sx)/ (num * 1.0));
 
      printf("\nGPU code is O(N^slope)");
      printf("\nSlope estimates are :");
      for (i=0; i<num-1; i++){
        printf(" %lf",slope[i]);
      }
      printf("\nSlope estimate for line fit is: %lf\n", b2);
	  
	  if (b2 < 1.2) grade = 40.00;
	  else if (b2 < 1.4) grade = (1.4-b2)/(0.2) * 40.00;
	  else grade =0.0;
	  
      for (i=0; i<=num;i++) {
         ss[i] = t[i+num]/t[i];
      }
 
      printf("\nGPU speedup estimates are :\n");
      for (i=0; i<num; i++){
        printf(" %7.2lf",ss[i]);
      }
      printf(" (speedup)\n");

	ss_avg=0.0;
	  for (i=0; i<num; i++){
        ss_avg+=ss[i];
      }
	  ss_avg/=num;
	  
	  printf("Average speedup: %7.2lf \n\n",ss_avg);

	  if (ss_avg > 9) ssgrade=60.00;
	  else if (ss_avg > 5) ssgrade=40.00 + 20.00 * (ss_avg-5)/4;
	  else if (ss_avg > 3) ssgrade= (ss_avg-3)/2 * 40.00;
	  else ssgrade =0;
	  
	if (grade > 0)
	  grade= grade + ssgrade;
	  
	  printf("\nGPU Total Grade = %7.2f\n\n",grade);
    }

    fclose( fread );
    
    return 0;
}
